#include "hip/hip_runtime.h"
#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>
#include <thrust/partition.h>
#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "material.h"
#include "intersections.h"
#include "interactions.h"
#include "sampler.h"
#include "macro.h"


//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::vec3 color;
		color = image[index] / (float)iter;
		//color = uncharted2filmic(color);
        color = ACES(color);
        glm::ivec3 icolor = glm::clamp(glm::ivec3(color * 255.f), glm::ivec3(0), glm::ivec3(255));
        // Each thread writes one pixel location in the texture (textel)

        pbo[index].w = 0;
        pbo[index].x = icolor.x;
        pbo[index].y = icolor.y;
        pbo[index].z = icolor.z;
    }
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...
int* dev_material_ids = NULL;
thrust::device_ptr<int> dev_thrust_material_ids = NULL;
thrust::device_ptr<PathSegment> dev_thrust_paths;
thrust::device_ptr<ShadeableIntersection> dev_thrust_intersections;
void InitDataContainer(GuiDataContainer* imGuiData)
{
    guiData = imGuiData;
}

void pathtraceInit(Scene* scene)
{
    hst_scene = scene;

    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // TODO: initialize any extra device memeory you need
	hipMalloc(&dev_material_ids, pixelcount * sizeof(int));
	dev_thrust_material_ids = thrust::device_pointer_cast(dev_material_ids);
	dev_thrust_paths = thrust::device_pointer_cast(dev_paths);
	dev_thrust_intersections = thrust::device_pointer_cast(dev_intersections);
    checkCUDAError("pathtraceInit");
}

void pathtraceFree()
{
    hipFree(dev_image);  // no-op if dev_image is null
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);
    // TODO: clean up any extra device memory you created
    hipFree(dev_material_ids);
    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
GPU Ray physical_light(const Camera& cam, int x, int y, thrust::default_random_engine &rng) {
    Ray ray;
    glm::vec4 r = sample4D(rng);
    float aspect = float(cam.resolution.x) / cam.resolution.y;
    float tanFovY = glm::tan(glm::radians(cam.fov.y));
    glm::vec2 pixelSize = 1.f / glm::vec2(cam.resolution);
    glm::vec2 scr = glm::vec2(x, y) * pixelSize;
    glm::vec2 ruv = scr + pixelSize * glm::vec2(r.x, r.y);
    ruv = 1.f - ruv * 2.f;

    glm::vec3 pLens = glm::vec3(squareToDiskConcentric(glm::vec2(r.z, r.w)) * cam.lensRadius, 0.f);
    glm::vec3 pFocusPlane = glm::vec3(ruv * glm::vec2(aspect, 1.f) * tanFovY, 1.f) * cam.focalDist;
    glm::vec3 dir = pFocusPlane - pLens;
    ray.direction = glm::normalize(glm::mat3(cam.right, cam.up, cam.view) * dir);
    ray.origin = cam.position + cam.right * pLens.x + cam.up * pLens.y;
    return ray;
}
GPU  glm::vec3 random_light_dir(const Camera& cam, int x, int y, thrust::default_random_engine & rng) {
    thrust::uniform_real_distribution<float> u01(-0.5f, 0.5f);
    float jitterX = u01(rng);
    float jitterY = u01(rng);
    return glm::normalize(cam.view
        - cam.right * cam.pixelLength.x * ((float)x + jitterX - (float)cam.resolution.x * 0.5f)
        - cam.up * cam.pixelLength.y * ((float)y + jitterY - (float)cam.resolution.y * 0.5f)
    );
}
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment& segment = pathSegments[index];


        // TODO: implement antialiasing by jittering the ray
		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
        
        //segment.ray.direction = random_light_dir(cam, x, y, rng);
        segment.ray = physical_light(cam, x, y, rng);
        //segment.ray.origin = cam.position;
        segment.radiance = glm::vec3(0.f);
        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);
        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
    }
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersectionsScene(
    int depth,
    int num_paths,
    PathSegment* pathSegments,
    GPUScene* dev_scene,
    ShadeableIntersection* intersections)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;
    
   
    if (path_index >= num_paths) {
        return;
    }
    PathSegment pathSegment = pathSegments[path_index];
    ShadeableIntersection isect;
    dev_scene->intersectTest(pathSegment.ray, isect);
    
    if (isect.primitiveId != -1) {
        if (dev_scene->materials[isect.materialId].type == MaterialType::Light) {
            if (depth != 0) {
                isect.prev = pathSegment.ray.origin;
            }
        }
        else {
            isect.dir = -pathSegment.ray.direction;
        }
    }
    intersections[path_index] = isect;
}
// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.

__global__ void pathIntegrator(
    int iter,
    int depth,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    GPUScene *dev_scene) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_paths) {
        return;
    }
    ShadeableIntersection intersection = shadeableIntersections[idx];
    PathSegment& segment = pathSegments[idx];
    if (intersection.t > 0.f) {
        thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
	    Material material = dev_scene->materials[intersection.materialId];
        segment.ray.origin = intersection.point;
        if (material.type == MaterialType::Light) {
            segment.color *= (material.color * material.emittance);
		    segment.remainingBounces = 0;
	    }
        else {
            BSDFSample sampler;
            material.SampleBSDF(intersection.surfaceNormal, intersection.dir, sample3D(rng), sampler);
            if (sampler.pdf < 0 || sampler.flags == Unset) {
                segment.remainingBounces = 0;
            }
            else {
                bool isDelta = sampler.flags & BxDFFlags::Specular;
                segment.color *= sampler.bsdf / sampler.pdf;
                if(!isDelta)
				    segment.color *= glm::dot(sampler.wi, intersection.surfaceNormal);
                segment.ray.direction = glm::normalize(sampler.wi);
			    segment.remainingBounces--;
            }
        }
    }
    else {
		segment.color = glm::vec3(0.0f);
		segment.remainingBounces = 0;
    }
	segment.ray.origin += EPSILON * segment.ray.direction;
}

__global__ void misPathIntegrator(
    int iter,
    int depth,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    GPUScene* dev_scene) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_paths) {
        return;
    }
	ShadeableIntersection intersection = shadeableIntersections[idx];
	PathSegment& segment = pathSegments[idx];
    if (intersection.primitiveId == -1) {
        return;
    }
	thrust::default_random_engine rng(makeSeededRandomEngine(iter, idx, depth));
	Material material = dev_scene->materials[intersection.materialId];
    glm::vec3 sumRadiance(0.f);
    if (material.type == MaterialType::Light) {
		glm::vec3 radiance = material.color * material.emittance;
        if (depth == 0) {
			sumRadiance += radiance;
        }
        else if (segment.deltaSample) {
			sumRadiance += radiance * segment.color;
        }
        else {
            float lightPdf = luminance(radiance) * dev_scene->devSumLightPowerInv *
                computeSolidAngle(intersection.prev, intersection.point, intersection.surfaceNormal);
            float bsdfPdf = segment.pdf;
			float weight = powerHeuristic(bsdfPdf, lightPdf);
			sumRadiance += radiance * segment.color * weight;
        }
		segment.remainingBounces = 0;
    }
    else {
		bool deltaBSDF = (material.type == MaterialType::Dielectric);
        if (!deltaBSDF) {
			glm::vec3 radiance(0.f);
            glm::vec3 wi;
			float lightPdf = dev_scene->sampleDirectLight(intersection.prev, sample4D(rng), radiance, wi);
            if (lightPdf > 0) {
				glm::vec3 bsdf = material.BSDF(intersection.surfaceNormal, intersection.prev, wi);
				float bsdfPdf = material.pdf(intersection.surfaceNormal, intersection.prev, wi);
				float weight = powerHeuristic(lightPdf, bsdfPdf);
                sumRadiance += segment.color * bsdf * radiance * weight * max(0.f, glm::dot(intersection.surfaceNormal, wi)) / lightPdf;
            }
        }
		BSDFSample sample;
		material.SampleBSDF(intersection.surfaceNormal, intersection.prev, sample3D(rng), sample);
        if (sample.flags != Unset) {
			bool deltaSample = sample.flags & BxDFFlags::Specular;
			segment.color *= sample.bsdf / sample.pdf;
			if (!deltaSample)
                segment.color *= glm::abs(glm::dot(sample.wi, intersection.surfaceNormal));
			segment.ray.direction = glm::normalize(sample.wi);
			segment.ray.origin = intersection.point + EPSILON * segment.ray.direction;
			segment.deltaSample = deltaSample;
			segment.pdf = sample.pdf;
			segment.remainingBounces--;
        }
        else {
			segment.remainingBounces = 0;
        }
    }
    segment.radiance += sumRadiance;
}
// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        image[iterationPath.pixelIndex] += iterationPath.color;
    }
}
struct IsPathRunning {
    CPUGPU bool operator()(const PathSegment& path) const {
        return path.remainingBounces != 0;
    }
};
struct sortByMaterial {
    CPUGPU bool operator()(
        const ShadeableIntersection& a, 
        const ShadeableIntersection& b) {
        return a.materialId < b.materialId;
    }
};
/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter)
{
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing

    generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
    checkCUDAError("generate camera ray");

    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount;
    int num_paths = dev_path_end - dev_paths;

    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks

    bool iterationComplete = false;
    while (!iterationComplete)
    {
        // clean shading chunks
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

        // tracing
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        /*computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>> (
            depth,
            num_paths,
            dev_paths,
            dev_geoms,
            hst_scene->geoms.size(),
            dev_intersections
        );*/
        computeIntersectionsScene << <numblocksPathSegmentTracing, blockSize1d >> > (
            depth,
            num_paths,
            dev_paths,
			hst_scene->devScene,
			dev_intersections
			);
        checkCUDAError("trace one bounce");
        hipDeviceSynchronize();
        depth++;

        // TODO:
        // --- Shading Stage ---
        // Shade path segments based on intersections and generate new rays by
        // evaluating the BSDF.
        // Start off with just a big kernel that handles all the different
        // materials you have in the scenefile.
        // TODO: compare between directly shading the path segments and shading
        // path segments that have been reshuffled to be contiguous in memory.

        // reshuffle material
        if (RESUFFLE_BY_MATERIAL) {
            thrust::sort_by_key(
                thrust::device_pointer_cast(dev_intersections),
                thrust::device_pointer_cast(dev_intersections + num_paths),
                thrust::make_zip_iterator(
                    thrust::make_tuple(
                        thrust::device_pointer_cast(dev_paths),
                        thrust::device_pointer_cast(dev_intersections)
                    )
                ),
                sortByMaterial()
            
            );
        }
		pathIntegrator << <numblocksPathSegmentTracing, blockSize1d >> > (
			iter,
            depth,
			num_paths,
			dev_intersections,
			dev_paths,
			hst_scene->devScene
			);
		//misPathIntegrator << <numblocksPathSegmentTracing, blockSize1d >> > (
		//	iter,
		//	depth,
		//	num_paths,
		//	dev_intersections,
		//	dev_paths,
		//	hst_scene->devScene
		//	);
        checkCUDAError("shading");
        hipDeviceSynchronize();
        PathSegment* new_end = thrust::stable_partition(
            thrust::device,
            dev_paths,
            dev_paths + num_paths,
            IsPathRunning()
        );
        hipDeviceSynchronize();
        // iterationComplete = true; // TODO: should be based off stream compaction results.
        int existing_paths = new_end - dev_paths;
        num_paths = existing_paths;
        iterationComplete = (num_paths == 0) || (depth >= traceDepth);
        if (guiData != NULL)
        {
            guiData->TracedDepth = depth;
        }
    }

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather<<<numBlocksPixels, blockSize1d>>>(pixelcount, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
